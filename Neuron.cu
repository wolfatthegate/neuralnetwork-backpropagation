#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

enum neuralNetworkLayerType { inputLayer = 0, hiddenLayer = 1, outputLayer = 2 }; 

double applyActivationFunction(double weightedSum) {
	// activation function is a sigmoid function
	return (1.0 / (1 + exp(-1.0 * weightedSum)));  
}

double derivative(double output) {
	return output * (1.0 - output); 
}

struct neuron {
	
	double threshold = 0; 
	double weight[2] = {0, 0}; 
	double output = 0; 
	double error = 0; 
	neuralNetworkLayerType layerType; 
};

void forwardProp(double input[], struct neuron *neurons) {
	double weightedSum = 0; 
	for( int i = 0; i < (int) sizeof (neurons); i++){
		switch (neurons[i].layerType) {
			case 0: // input layer
				neurons[i].output = input[i];  
				break;
			case 1: // hidden layer
				weightedSum = neurons[i].threshold + 
								  neurons[i].weight[0] * neurons[0].output + 
		    		              neurons[i].weight[1] * neurons[1].output;
				neurons[i].output = applyActivationFunction(weightedSum); 
				break; 
			case 2: // output layer
				weightedSum = neurons[i].threshold + 
	    		                  neurons[i].weight[0] * neurons[2].output + 
	    		                  neurons[i].weight[1] * neurons[3].output;
		    	neurons[i].output = applyActivationFunction(weightedSum); 
				break; 
		}
	}
}

void backpropError(double targetResult, struct neuron *neurons){
	// calculating for output neurons
	neurons[4].error = (targetResult - neurons[4].output) * derivative(neurons[4].output);
	neurons[4].threshold = neurons[4].threshold + LEARNING_RATE * neurons[4].error;
	neurons[4].weight[0] = neurons[4].weight[0] + LEARNING_RATE * neurons[4].error * neurons[2].output; 
	neurons[4].weight[1] = neurons[4].weight[1] + LEARNING_RATE * neurons[4].error * neurons[3].output; 
	
	// calculating for hidden layer 1 
	neurons[3].error = (neurons[4].weight[1] * neurons[4].error) * derivative(neurons[3].output);
	neurons[3].threshold = neurons[3].threshold + LEARNING_RATE * neurons[3].error;
	neurons[3].weight[0] = neurons[3].weight[0] + LEARNING_RATE * neurons[3].error * neurons[0].output;
	neurons[3].weight[1] = neurons[3].weight[1] + LEARNING_RATE * neurons[3].error * neurons[1].output;

	// calculating for hidden layer 2 
	neurons[2].error = (neurons[4].weight[0] * neurons[4].error) * derivative(neurons[2].output);
	neurons[2].threshold = neurons[2].threshold + LEARNING_RATE * neurons[2].error;
	neurons[2].weight[0] = neurons[2].weight[0] + LEARNING_RATE * neurons[2].error * neurons[0].output;
	neurons[2].weight[1] = neurons[2].weight[1] + LEARNING_RATE * neurons[2].error * neurons[1].output;

}

void setNeurons(struct neuron *neurons){

	srand((long)time(NULL)); /* initialize rand() */
	for (int i = 0; i < 2; i ++){
		neurons[i].threshold = 0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].weight[0] =  0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].weight[1] =  0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].layerType = inputLayer; 
	}

	for (int i = 2; i < 4; i ++){
		neurons[i].threshold = 0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].weight[0] =  0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].weight[1] =  0.5 - (rand()/(double)RAND_MAX); 
		neurons[i].layerType = hiddenLayer; 
	}

	neurons[4].threshold = 0.5 - (rand()/(double)RAND_MAX); 
	neurons[4].weight[0] =  0.5 - (rand()/(double)RAND_MAX); 
	neurons[4].weight[1] =  0.5 - (rand()/(double)RAND_MAX); 
	neurons[4].layerType = outputLayer; 
}

void printTrainingData(struct neuron *neurons){
	
	printf("[(I: %.2f), (I: %.2f), ", neurons[0].output, neurons[1].output); 
	printf("(H: %.2f, %.2f, %.2f, %.5f), ", neurons[2].weight[0], neurons[2].weight[1], neurons[2].threshold, neurons[2].output);
	printf("(H: %.2f, %.2f, %.2f, %.5f), ", neurons[3].weight[0], neurons[3].weight[1], neurons[3].threshold, neurons[3].output);
	printf("(O: %.2f, %.2f, %.2f, %.5f)]\n ", neurons[4].weight[0], neurons[4].weight[1], neurons[4].threshold, neurons[4].output);
}

void printResult(double result[]) {
	printf("    Input 1    |    Input 2    | Target Result |  Result    \n");
	printf("-------------------------------------------------------------\n");
	for(int i = 0; i < 4; i++ ) {
		for(int j = 0; j < 2; j++) {
			printf("    %.5f    |", TRAINING_DATA[i][0][j]); 
		}
		printf("    %.5f    |   %.5f   \n", TRAINING_DATA[i][1][0], result[i]);
	}
}
