/* 
 * CPU version of BackPropagation Neural Network written in CUDA. One can change the extension .cu to .c 
 * and compile the program with C compiler. 
 * 
 * This program is a rework of Source code for Neural Networks w/ JAVA (Tutorial 09) - Backpropagation 01
 * from http://zaneacademy.com
 * 
 * Author - Waylon Luo
 * Date - April 23, 2020 
 *
 */

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <algorithm>
#include <vector>

#define LEARNING_RATE 0.80 
#define NUMB_OF_EPOCHS 100
#define TD_X 4 // training data in x- dimension
#define TD_Y 2 // training data in y- dimension
#define TD_Z 2 // training data in z- dimension   

double rand_double();
double TRAINING_DATA[TD_X][TD_Y][TD_Z] = {{{0,0},{0}},
					          	 {{0,1},{1}},
					          	 {{1,0},{1}},
					          	 {{1,1},{0}}}; 

#include "Neuron.cu"

int main(void){
 
 	double result[] = {0, 0, 0, 0}; 
 	// declare and initialize neurons 
	struct neuron neurons[5]; 
	setNeurons(neurons);

	// forward propagation before the training
	for(int i = 0; i < TD_X; i++) {   // TD_X - Traning Data Dimension X 
		forwardProp(TRAINING_DATA[i][0], neurons);
		result[i] = neurons[4].output; // get output
	}
	printResult(result); 

	// training 100 * 100 = 10,000 trainings 
	for(int x = 0; x < 100; x++){
		for(int i = 0; i < NUMB_OF_EPOCHS; i++) {   
			if(i%100 == 0) {
				printf("[epoch %d ]\n", i);
			}
			for(int j = 0; j < TD_X; j++) {  // TD_X - Traning Data Dimension X 
				forwardProp(TRAINING_DATA[j][0], neurons);
				backpropError(TRAINING_DATA[j][1][0], neurons);		
				if(i%100 == 0) printTrainingData(neurons); 
			}
		}
	}
	printf("[done training]\n");

	// forward propagation after the training
	for(int i = 0; i < TD_X; i++) {
		forwardProp(TRAINING_DATA[i][0], neurons);
		result[i] = neurons[4].output; // get output
	}
	printResult(result); 

	return(1);
}